#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// SPDX-FileCopyrightText: 2021 Brian Hamilton

#include "engine_gpu.hpp"

#include "pffdtd/assert.hpp"
#include "pffdtd/config.hpp"
#include "pffdtd/progress.hpp"
#include "pffdtd/utility.hpp"

#include <cmath>
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <vector>

namespace pffdtd {

// want 0 to map to 1, otherwise kernel errors
constexpr auto CU_DIV_CEIL(auto x, auto y) { return ((DIV_CEIL(x, y) == 0) ? (1) : (DIV_CEIL(x, y))); }

// thread-block dims for 3d kernels
constexpr auto cuBx = 32;
constexpr auto cuBy = 2;
constexpr auto cuBz = 2;

// thread-block dims for 2d kernels (fcc fold, ABCs)
constexpr auto cuBx2 = 16;
constexpr auto cuBy2 = 8;

// thread-block dims for 1d kernels (bn, ABC loss)
constexpr auto cuBrw = 128;
constexpr auto cuBb  = 128;

// NOLINTBEGIN(cppcoreguidelines-avoid-non-const-global-variables)
// constant memory (all per device)
__constant__ Real c1;
__constant__ Real c2;
__constant__ Real cl;
__constant__ Real csl2;
__constant__ Real clo2;
__constant__ int64_t cuNx;
__constant__ int64_t cuNy;
__constant__ int64_t cuNz;
__constant__ int64_t cuNb;
__constant__ int64_t cuNbl;
__constant__ int64_t cuNba;
__constant__ int64_t cuNxNy;
__constant__ int8_t cuMb[MNm]; // to store Mb per mat

// NOLINTEND(cppcoreguidelines-avoid-non-const-global-variables)

// this is data on host, sometimes copied and recomputed for copy to GPU devices
// (indices), sometimes just aliased pointers (scalar arrays)
template<typename Float>
struct HostData {      // arrays on host (for copy), mirrors gpu local data
  double* in_sigs{};   // aliased
  Float* u_out_buf{};  // aliased
  double* u_out{};     // aliased
  Float* ssaf_bnl{};   // aliased
  int64_t* in_ixyz{};  // recomputed
  int64_t* out_ixyz{}; // recomputed
  int64_t* bn_ixyz{};  // recomputed
  int64_t* bnl_ixyz{}; // recomputed
  int64_t* bna_ixyz{}; // recomputed
  int8_t* Q_bna{};     // aliased
  uint16_t* adj_bn{};  // aliased
  int8_t* mat_bnl{};   // aliased
  uint8_t* bn_mask{};  // recomputed
  int8_t* K_bn{};      // aliased
  int64_t Ns{};
  int64_t Nr{};
  int64_t Npts{};
  int64_t Nx{};
  int64_t Nxh{};
  int64_t Nb{};
  int64_t Nbl{};
  int64_t Nba{};
  int64_t Nbm{}; // bytes for bn_mask
};

// these are arrays pointing to GPU device memory, or CUDA stuff (dim3, events)
template<typename Float>
struct DeviceData { // for or on gpu (arrays all on GPU)
  int64_t* bn_ixyz{};
  int64_t* bnl_ixyz{};
  int64_t* bna_ixyz{};
  int8_t* Q_bna{};
  int64_t* out_ixyz{};
  uint16_t* adj_bn{};
  Float* ssaf_bnl{};
  uint8_t* bn_mask{};
  int8_t* mat_bnl{};
  int8_t* K_bn{};
  Float* mat_beta{};
  MatQuad<Float>* mat_quads{};
  Float* u0{};
  Float* u1{};
  Float* u0b{};
  Float* u1b{};
  Float* u2b{};
  Float* u2ba{};
  Float* vh1{};
  Float* gh1{};
  Float* u_out_buf{};
  dim3 block_dim_air;
  dim3 grid_dim_air;
  dim3 block_dim_fold;
  dim3 grid_dim_fold;
  dim3 block_dim_readout;
  dim3 grid_dim_readout;
  dim3 block_dim_bn;
  dim3 block_dim_halo_xy;
  dim3 block_dim_halo_yz;
  dim3 block_dim_halo_xz;
  dim3 grid_dim_bn;
  dim3 grid_dim_bnl;
  dim3 grid_dim_bna;
  dim3 grid_dim_halo_xy;
  dim3 grid_dim_halo_yz;
  dim3 grid_dim_halo_xz;
  hipStream_t cuStream_air{};
  hipStream_t cuStream_bn{};
  hipEvent_t cuEv_air_start{};
  hipEvent_t cuEv_air_end{};
  hipEvent_t cuEv_bn_roundtrip_start{};
  hipEvent_t cuEv_bn_roundtrip_end{};
  hipEvent_t cuEv_readout_end{};
  int64_t totalmembytes{};
};

// NB. 'x' is contiguous dim in CUDA domain

// vanilla scheme, unrolled, intrinsics to control rounding errors
template<typename Float>
__global__ void
KernelAirCart(Float* __restrict__ u0, Float const* __restrict__ u1, uint8_t const* __restrict__ bn_mask) {
  int64_t const cx = blockIdx.x * cuBx + threadIdx.x + 1;
  int64_t const cy = blockIdx.y * cuBy + threadIdx.y + 1;
  int64_t const cz = blockIdx.z * cuBz + threadIdx.z + 1;
  if ((cx < cuNx - 1) && (cy < cuNy - 1) && (cz < cuNz - 1)) {
    int64_t const ii = cz * cuNxNy + cy * cuNx + cx;
    // divide-conquer add for better accuracy
    Float tmp1 = NAN;
    Float tmp2 = NAN;
    tmp1       = ADD_O(u1[ii + cuNxNy], u1[ii - cuNxNy]);
    tmp2       = ADD_O(u1[ii + cuNx], u1[ii - cuNx]);
    tmp1       = ADD_O(tmp1, tmp2);
    tmp2       = ADD_O(u1[ii + 1], u1[ii - 1]);
    tmp1       = ADD_O(tmp1, tmp2);
    tmp1       = FMA_D(c1, u1[ii], FMA_D(c2, tmp1, -u0[ii]));

    // write final value back to global memory
    if ((GET_BIT(bn_mask[ii >> 3], ii % 8)) == 0) {
      u0[ii] = tmp1;
    }
  }
}

// air update for FCC, on folded grid (improvement to 2013 DAFx paper)
template<typename Float>
__global__ void
KernelAirFCC(Float* __restrict__ u0, Float const* __restrict__ u1, uint8_t const* __restrict__ bn_mask) {
  // get ix,iy,iz from thread and block Id's
  int64_t const cx = blockIdx.x * cuBx + threadIdx.x + 1;
  int64_t const cy = blockIdx.y * cuBy + threadIdx.y + 1;
  int64_t const cz = blockIdx.z * cuBz + threadIdx.z + 1;
  if ((cx < cuNx - 1) && (cy < cuNy - 1) && (cz < cuNz - 1)) {
    // x is contiguous
    int64_t const ii = cz * cuNxNy + cy * cuNx + cx;
    Float tmp1       = NAN;
    Float tmp2       = NAN;
    Float tmp3       = NAN;
    Float tmp4       = NAN;
    // divide-conquer add as much as possible
    tmp1 = ADD_O(u1[ii + cuNxNy + cuNx], u1[ii - cuNxNy - cuNx]);
    tmp2 = ADD_O(u1[ii + cuNx + 1], u1[ii - cuNx - 1]);
    tmp1 = ADD_O(tmp1, tmp2);
    tmp3 = ADD_O(u1[ii + cuNxNy + 1], u1[ii - cuNxNy - 1]);
    tmp4 = ADD_O(u1[ii + cuNxNy - cuNx], u1[ii - cuNxNy + cuNx]);
    tmp3 = ADD_O(tmp3, tmp4);
    tmp2 = ADD_O(u1[ii + cuNx - 1], u1[ii - cuNx + 1]);
    tmp1 = ADD_O(tmp1, tmp2);
    tmp4 = ADD_O(u1[ii + cuNxNy - 1], u1[ii - cuNxNy + 1]);
    tmp3 = ADD_O(tmp3, tmp4);
    tmp1 = ADD_O(tmp1, tmp3);
    tmp1 = FMA_D(c1, u1[ii], FMA_D(c2, tmp1, -u0[ii]));
    // write final value back to global memory
    if ((GET_BIT(bn_mask[ii >> 3], ii % 8)) == 0) {
      u0[ii] = tmp1;
    }
  }
}

// this folds in half of FCC subgrid so everything is nicely homogenous (no
// braching for stencil)
template<typename Float>
__global__ void KernelFoldFCC(Float* __restrict__ u1) {
  int64_t const cx = blockIdx.x * cuBx2 + threadIdx.x;
  int64_t const cz = blockIdx.y * cuBy2 + threadIdx.y;
  // fold is along middle dimension
  if ((cx < cuNx) && (cz < cuNz)) {
    u1[cz * cuNxNy + (cuNy - 1) * cuNx + cx] = u1[cz * cuNxNy + (cuNy - 2) * cuNx + cx];
  }
}

// rigid boundaries, cartesian, using adj info
template<typename Float>
__global__ void KernelBoundaryRigidCart(
    Float* __restrict__ u0,
    Float const* __restrict__ u1,
    uint16_t const* __restrict__ adj_bn,
    int64_t const* __restrict__ bn_ixyz,
    int8_t const* __restrict__ K_bn
) {
  int64_t const nb = blockIdx.x * cuBb + threadIdx.x;
  if (nb < cuNb) {
    int64_t const ii   = bn_ixyz[nb];
    uint16_t const adj = adj_bn[nb];
    Float const K      = K_bn[nb];

    Float const _2 = 2.0;
    Float const b1 = (_2 - csl2 * K);
    Float const b2 = c2;

    Float tmp1 = NAN;
    Float tmp2 = NAN;
    tmp1       = ADD_O((Float)GET_BIT(adj, 0) * u1[ii + cuNxNy], (Float)GET_BIT(adj, 1) * u1[ii - cuNxNy]);
    tmp2       = ADD_O((Float)GET_BIT(adj, 2) * u1[ii + cuNx], (Float)GET_BIT(adj, 3) * u1[ii - cuNx]);
    tmp1       = ADD_O(tmp1, tmp2);
    tmp2       = ADD_O((Float)GET_BIT(adj, 4) * u1[ii + 1], (Float)GET_BIT(adj, 5) * u1[ii - 1]);
    tmp1       = ADD_O(tmp1, tmp2);
    tmp1       = FMA_D(b1, u1[ii], FMA_D(b2, tmp1, -u0[ii]));

    // u0[ii] = partial; //write back to global memory
    u0[ii] = tmp1; // write back to global memory
  }
}

// rigid boundaries, FCC, using adj info
template<typename Float>
__global__ void KernelBoundaryRigidFCC(
    Float* __restrict__ u0,
    Float const* __restrict__ u1,
    uint16_t const* __restrict__ adj_bn,
    int64_t const* __restrict__ bn_ixyz,
    int8_t const* __restrict__ K_bn
) {
  int64_t const nb = blockIdx.x * cuBb + threadIdx.x;
  if (nb < cuNb) {
    int64_t const ii   = bn_ixyz[nb];
    uint16_t const adj = adj_bn[nb];
    Float const K      = K_bn[nb];

    Float const _2 = 2.0;
    Float const b1 = (_2 - csl2 * K);
    Float const b2 = c2;

    Float tmp1 = NAN;
    Float tmp2 = NAN;
    Float tmp3 = NAN;
    Float tmp4 = NAN;
    tmp1 = ADD_O((Float)GET_BIT(adj, 0) * u1[ii + cuNxNy + cuNx], (Float)GET_BIT(adj, 1) * u1[ii - cuNxNy - cuNx]);
    tmp2 = ADD_O((Float)GET_BIT(adj, 2) * u1[ii + cuNx + 1], (Float)GET_BIT(adj, 3) * u1[ii - cuNx - 1]);
    tmp1 = ADD_O(tmp1, tmp2);
    tmp3 = ADD_O((Float)GET_BIT(adj, 4) * u1[ii + cuNxNy + 1], (Float)GET_BIT(adj, 5) * u1[ii - cuNxNy - 1]);
    tmp4 = ADD_O((Float)GET_BIT(adj, 6) * u1[ii + cuNxNy - cuNx], (Float)GET_BIT(adj, 7) * u1[ii - cuNxNy + cuNx]);
    tmp3 = ADD_O(tmp3, tmp4);
    tmp2 = ADD_O((Float)GET_BIT(adj, 8) * u1[ii + cuNx - 1], (Float)GET_BIT(adj, 9) * u1[ii - cuNx + 1]);
    tmp1 = ADD_O(tmp1, tmp2);
    tmp4 = ADD_O((Float)GET_BIT(adj, 10) * u1[ii + cuNxNy - 1], (Float)GET_BIT(adj, 11) * u1[ii - cuNxNy + 1]);
    tmp3 = ADD_O(tmp3, tmp4);
    tmp1 = ADD_O(tmp1, tmp3);
    tmp1 = FMA_D(b1, u1[ii], FMA_D(b2, tmp1, -u0[ii]));

    u0[ii] = tmp1; // write back to global memory
  }
}

// ABC loss at boundaries of simulation grid
template<typename Float>
__global__ void KernelBoundaryABC(
    Float* __restrict__ u0,
    Float const* __restrict__ u2ba,
    int8_t const* __restrict__ Q_bna,
    int64_t const* __restrict__ bna_ixyz
) {
  int64_t const nb = blockIdx.x * cuBb + threadIdx.x;
  if (nb < cuNba) {
    Float const _1   = 1.0;
    Float const lQ   = cl * Q_bna[nb];
    int64_t const ib = bna_ixyz[nb];
    Float partial    = u0[ib];
    partial          = (partial + lQ * u2ba[nb]) / (_1 + lQ);
    u0[ib]           = partial;
  }
}

// Part of freq-dep boundary update
template<typename Float>
__global__ void KernelBoundaryFD(
    Float* __restrict__ u0b,
    Float const* u2b,
    Float* __restrict__ vh1,
    Float* __restrict__ gh1,
    Float const* ssaf_bnl,
    int8_t const* mat_bnl,
    Float const* __restrict__ mat_beta,
    MatQuad<Float> const* __restrict__ mat_quads
) {
  int64_t const nb = blockIdx.x * cuBb + threadIdx.x;
  if (nb < cuNbl) {
    Float const _1     = 1.0;
    Float const _2     = 2.0;
    int32_t const k    = mat_bnl[nb];
    Float const ssaf   = ssaf_bnl[nb];
    Float const lo2Kbg = clo2 * ssaf * mat_beta[k];
    Float const fac    = _2 * clo2 * ssaf / (_1 + lo2Kbg);

    Float u0bint       = u0b[nb];
    Float const u2bint = u2b[nb];

    u0bint = (u0bint + lo2Kbg * u2bint) / (_1 + lo2Kbg);

    Float vh1int[MMb]; // size has to be constant at compile time
    Float gh1int[MMb];
    for (int8_t m = 0; m < cuMb[k]; m++) { // faster on average than MMb
      int64_t const nbm        = m * cuNbl + nb;
      int32_t const mbk        = k * MMb + m;
      MatQuad<Float> const* tm = nullptr;
      tm                       = &(mat_quads[mbk]);
      vh1int[m]                = vh1[nbm];
      gh1int[m]                = gh1[nbm];
      u0bint -= fac * (_2 * (tm->bDh) * vh1int[m] - (tm->bFh) * gh1int[m]);
    }

    Float const du = u0bint - u2bint;

    // NOLINTBEGIN(clang-analyzer-core.UndefinedBinaryOperatorResult)
    for (int8_t m = 0; m < cuMb[k]; m++) { // faster on average than MMb
      int64_t const nbm        = m * cuNbl + nb;
      int32_t const mbk        = k * MMb + m;
      MatQuad<Float> const* tm = nullptr;
      tm                       = &(mat_quads[mbk]);
      Float const vh0m         = (tm->b) * du + (tm->bd) * vh1int[m] - _2 * (tm->bFh) * gh1int[m];
      gh1[nbm]                 = gh1int[m] + (vh0m + vh1int[m]) / _2;
      vh1[nbm]                 = vh0m;
    }
    // NOLINTEND(clang-analyzer-core.UndefinedBinaryOperatorResult)
    u0b[nb] = u0bint;
  }
}

// add source input (one at a time for simplicity)
template<typename Float>
__global__ void AddIn(Float* u0, Float sample) {
  u0[0] += sample;
}

// dst-src copy from buffer to grid
template<typename Float>
__global__ void CopyToGridKernel(Float* u, Float const* buffer, int64_t const* locs, int64_t N) {
  int64_t const i = blockIdx.x * cuBrw + threadIdx.x;
  if (i < N) {
    u[locs[i]] = buffer[i];
  }
}

// dst-src copy to buffer from  grid (not needed, but to make more explicit)
template<typename Float>
__global__ void CopyFromGridKernel(Float* buffer, Float const* u, int64_t const* locs, int64_t N) {
  int64_t const i = blockIdx.x * cuBrw + threadIdx.x;
  if (i < N) {
    buffer[i] = u[locs[i]];
  }
}

// flip halos for ABCs
template<typename Float>
__global__ void FlipHaloXY_Zbeg(Float* __restrict__ u1) {
  int64_t const cx = blockIdx.x * cuBx2 + threadIdx.x;
  int64_t const cy = blockIdx.y * cuBy2 + threadIdx.y;
  if ((cx < cuNx) && (cy < cuNy)) {
    int64_t ii = 0;
    ii         = 0 * cuNxNy + cy * cuNx + cx;
    u1[ii]     = u1[ii + 2 * cuNxNy];
  }
}

template<typename Float>
__global__ void FlipHaloXY_Zend(Float* __restrict__ u1) {
  int64_t const cx = blockIdx.x * cuBx2 + threadIdx.x;
  int64_t const cy = blockIdx.y * cuBy2 + threadIdx.y;
  if ((cx < cuNx) && (cy < cuNy)) {
    int64_t ii = 0;
    ii         = (cuNz - 1) * cuNxNy + cy * cuNx + cx;
    u1[ii]     = u1[ii - 2 * cuNxNy];
  }
}

template<typename Float>
__global__ void FlipHaloXZ_Ybeg(Float* __restrict__ u1) {
  int64_t const cx = blockIdx.x * cuBx2 + threadIdx.x;
  int64_t const cz = blockIdx.y * cuBy2 + threadIdx.y;
  if ((cx < cuNx) && (cz < cuNz)) {
    int64_t ii = 0;
    ii         = cz * cuNxNy + 0 * cuNx + cx;
    u1[ii]     = u1[ii + 2 * cuNx];
  }
}

template<typename Float>
__global__ void FlipHaloXZ_Yend(Float* __restrict__ u1) {
  int64_t const cx = blockIdx.x * cuBx2 + threadIdx.x;
  int64_t const cz = blockIdx.y * cuBy2 + threadIdx.y;
  if ((cx < cuNx) && (cz < cuNz)) {
    int64_t ii = 0;
    ii         = cz * cuNxNy + (cuNy - 1) * cuNx + cx;
    u1[ii]     = u1[ii - 2 * cuNx];
  }
}

template<typename Float>
__global__ void FlipHaloYZ_Xbeg(Float* __restrict__ u1) {
  int64_t const cy = blockIdx.x * cuBx2 + threadIdx.x;
  int64_t const cz = blockIdx.y * cuBy2 + threadIdx.y;
  if ((cy < cuNy) && (cz < cuNz)) {
    int64_t ii = 0;
    ii         = cz * cuNxNy + cy * cuNx + 0;
    u1[ii]     = u1[ii + 2];
  }
}

template<typename Float>
__global__ void FlipHaloYZ_Xend(Float* __restrict__ u1) {
  int64_t const cy = blockIdx.x * cuBx2 + threadIdx.x;
  int64_t const cz = blockIdx.y * cuBy2 + threadIdx.y;
  if ((cy < cuNy) && (cz < cuNz)) {
    int64_t ii = 0;
    ii         = cz * cuNxNy + cy * cuNx + (cuNx - 1);
    u1[ii]     = u1[ii - 2];
  }
}

// standard error checking
// NOLINTNEXTLINE
#define gpuErrchk(ans)                                                                                                 \
  { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char const* file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) {
      exit(code);
    }
  }
}

// print some device details
auto print_gpu_details(int i) -> uint64_t {
  hipDeviceProp_t prop{};
  hipGetDeviceProperties(&prop, i);
  printf("\nDevice Number: %d [%s]\n", i, prop.name);
  printf("  Compute: %d.%d\n", prop.major, prop.minor);
  printf("  Peak Memory Bandwidth: %.3f GB/s\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8.0) / 1.0e6);
  printf(
      "  Total global memory: [ %.3f GB | %.3f GiB | %lu MiB ]\n",
      (double)prop.totalGlobalMem / (1e9),
      (double)prop.totalGlobalMem / 1073741824ULL,
      prop.totalGlobalMem >> 20
  );
  printf("  Registers per block: %d\n", prop.regsPerBlock);
  printf("  Concurrent Kernels: %d\n", prop.concurrentKernels);
  printf("  Async Engine: %d\n", prop.asyncEngineCount);
  printf("\n");
  return prop.totalGlobalMem;
}

// input indices need to be sorted for multi-device allocation
void check_sorted(Simulation3D const* sim) {
  int64_t* bn_ixyz  = sim->bn_ixyz;
  int64_t* bnl_ixyz = sim->bnl_ixyz;
  int64_t* bna_ixyz = sim->bna_ixyz;
  int64_t* in_ixyz  = sim->in_ixyz;
  int64_t* out_ixyz = sim->out_ixyz;
  int64_t const Nb  = sim->Nb;
  int64_t const Nbl = sim->Nbl;
  int64_t const Nba = sim->Nba;
  int64_t const Ns  = sim->Ns;
  int64_t const Nr  = sim->Nr;
  for (int64_t i = 1; i < Nb; i++) {
    PFFDTD_ASSERT(bn_ixyz[i] > bn_ixyz[i - 1]); // check save_gpu_folder
  }
  for (int64_t i = 1; i < Nbl; i++) {
    PFFDTD_ASSERT(bnl_ixyz[i] > bnl_ixyz[i - 1]);
  }
  for (int64_t i = 1; i < Nba; i++) {
    PFFDTD_ASSERT(bna_ixyz[i] > bna_ixyz[i - 1]);
  }
  for (int64_t i = 1; i < Ns; i++) {
    PFFDTD_ASSERT(in_ixyz[i] > in_ixyz[i - 1]);
  }
  for (int64_t i = 1; i < Nr; i++) {
    PFFDTD_ASSERT(out_ixyz[i] >= out_ixyz[i - 1]); // possible to have duplicates
  }
}

// counts for splitting data across GPUs
void split_data(Simulation3D const* sim, HostData<Real>* ghds, int ngpus) {
  int64_t const Nx    = sim->Nx;
  int64_t const Ny    = sim->Ny;
  int64_t const Nz    = sim->Nz;
  HostData<Real>* ghd = nullptr;
  // initialise
  for (int gid = 0; gid < ngpus; gid++) {
    ghd      = &ghds[gid];
    ghd->Nx  = 0;
    ghd->Nb  = 0;
    ghd->Nbl = 0;
    ghd->Nba = 0;
    ghd->Ns  = 0;
    ghd->Nr  = 0;
  }

  // split Nx layers (Nz contiguous)
  int64_t const Nxm = Nx / ngpus;
  int64_t const Nxl = Nx % ngpus;

  for (int gid = 0; gid < ngpus; gid++) {
    ghd     = &ghds[gid];
    ghd->Nx = Nxm;
  }
  for (int gid = 0; gid < Nxl; gid++) {
    ghd = &ghds[gid];
    ghd->Nx += 1;
  }
  int64_t Nx_check = 0;
  for (int gid = 0; gid < ngpus; gid++) {
    ghd = &ghds[gid];
    printf("gid=%d, Nx[%d]=%ld, Nx=%ld\n", gid, gid, ghd->Nx, Nx);
    Nx_check += ghd->Nx;
  }
  PFFDTD_ASSERT(Nx_check == Nx);

  // now count Nr,Ns,Nb for each device
  auto Nxcc = std::vector<int64_t>(static_cast<size_t>(ngpus));
  Nxcc[0]   = ghds[0].Nx;
  printf("Nxcc[%d]=%ld\n", 0, Nxcc[0]);
  for (int gid = 1; gid < ngpus; gid++) {
    ghd       = &ghds[gid];
    Nxcc[gid] = ghd->Nx + Nxcc[gid - 1];
    printf("Nxcc[%d]=%ld\n", gid, Nxcc[gid]);
  }

  // bn_ixyz - Nb
  int64_t* bn_ixyz = sim->bn_ixyz;
  int64_t const Nb = sim->Nb;
  {
    int gid = 0;
    for (int64_t i = 0; i < Nb; i++) {
      while (bn_ixyz[i] >= Nxcc[gid] * Ny * Nz) {
        gid++;
      }
      (ghds[gid].Nb)++;
    }
  }
  int64_t Nb_check = 0;
  for (int gid = 0; gid < ngpus; gid++) {
    ghd = &ghds[gid];
    printf("gid=%d, Nb[%d]=%ld, Nb=%ld\n", gid, gid, ghd->Nb, Nb);
    Nb_check += ghd->Nb;
  }
  PFFDTD_ASSERT(Nb_check == Nb);

  // bnl_ixyz - Nbl
  int64_t* bnl_ixyz = sim->bnl_ixyz;
  int64_t const Nbl = sim->Nbl;
  {
    int gid = 0;
    for (int64_t i = 0; i < Nbl; i++) {
      while (bnl_ixyz[i] >= Nxcc[gid] * Ny * Nz) {
        gid++;
      }
      (ghds[gid].Nbl)++;
    }
  }
  int64_t Nbl_check = 0;
  for (int gid = 0; gid < ngpus; gid++) {
    ghd = &ghds[gid];
    printf("gid=%d, Nbl[%d]=%ld, Nbl=%ld\n", gid, gid, ghd->Nbl, Nbl);
    Nbl_check += ghd->Nbl;
  }
  PFFDTD_ASSERT(Nbl_check == Nbl);

  // bna_ixyz - Nba
  int64_t* bna_ixyz = sim->bna_ixyz;
  int64_t const Nba = sim->Nba;
  {
    int gid = 0;
    for (int64_t i = 0; i < Nba; i++) {
      while (bna_ixyz[i] >= Nxcc[gid] * Ny * Nz) {
        gid++;
      }
      (ghds[gid].Nba)++;
    }
  }
  int64_t Nba_check = 0;
  for (int gid = 0; gid < ngpus; gid++) {
    ghd = &ghds[gid];
    printf("gid=%d, Nba[%d]=%ld, Nbl=%ld\n", gid, gid, ghd->Nba, Nba);
    Nba_check += ghd->Nba;
  }
  PFFDTD_ASSERT(Nba_check == Nba);

  // in_ixyz - Ns
  int64_t* in_ixyz = sim->in_ixyz;
  int64_t const Ns = sim->Ns;
  {
    int gid = 0;
    for (int64_t i = 0; i < Ns; i++) {
      while (in_ixyz[i] >= Nxcc[gid] * Ny * Nz) {
        gid++;
      }
      (ghds[gid].Ns)++;
    }
  }
  int64_t Ns_check = 0;
  for (int gid = 0; gid < ngpus; gid++) {
    ghd = &ghds[gid];
    printf("gid=%d, Ns[%d]=%ld, Ns=%ld\n", gid, gid, ghd->Ns, Ns);
    Ns_check += ghd->Ns;
  }
  PFFDTD_ASSERT(Ns_check == Ns);

  // out_ixyz - Nr
  int64_t* out_ixyz = sim->out_ixyz;
  int64_t const Nr  = sim->Nr;
  {
    int gid = 0;
    for (int64_t i = 0; i < Nr; i++) {
      while (out_ixyz[i] >= Nxcc[gid] * Ny * Nz) {
        gid++;
      }
      (ghds[gid].Nr)++;
    }
  }
  int64_t Nr_check = 0;
  for (int gid = 0; gid < ngpus; gid++) {
    ghd = &ghds[gid];
    printf("gid=%d, Nr[%d]=%ld, Nr=%ld\n", gid, gid, ghd->Nr, Nr);
    Nr_check += ghd->Nr;
  }
  PFFDTD_ASSERT(Nr_check == Nr);
}

// run the sim!
auto run(Simulation3D const& sim) -> double {
  // if you want to test synchronous, env variable for that
  char const* s = getenv("CUDA_LAUNCH_BLOCKING");
  if (s != nullptr) {
    if (s[0] == '1') {
      printf("******************SYNCHRONOUS (DEBUG  ONLY!!!)*********************\n");
      printf("...continue?\n");
      getchar();
    }
  }

  PFFDTD_ASSERT((sim.fcc_flag != 1)); // uses either cartesian or FCC folded grid

  int ngpus     = 0;
  int max_ngpus = 0;
  hipGetDeviceCount(&max_ngpus); // control outside with CUDA_VISIBLE_DEVICES
  ngpus = max_ngpus;
  PFFDTD_ASSERT(ngpus < (sim.Nx));
  DeviceData<Real>* gds = nullptr;
  allocate_zeros((void**)&gds, ngpus * sizeof(DeviceData<Real>));
  HostData<Real>* ghds = nullptr;
  allocate_zeros((void**)&ghds, ngpus * sizeof(HostData<Real>)); // one bit per

  if (ngpus > 1) {
    check_sorted(&sim); // needs to be sorted for multi-GPU
  }

  // get local counts for Nx,Nb,Nr,Ns
  split_data(&sim, ghds, ngpus);

  for (int gid = 0; gid < ngpus; gid++) {
    gds[gid].totalmembytes = print_gpu_details(gid);
  }

  Real lo2 = sim.lo2;
  Real a1  = sim.a1;
  Real a2  = sim.a2;
  Real l   = sim.l;
  Real sl2 = sim.sl2;

  // timing stuff
  double time_elapsed             = 0.0;
  double time_elapsed_bn          = 0.0;
  double time_elapsed_sample      = NAN;
  double time_elapsed_sample_bn   = NAN;
  double time_elapsed_air         = 0.0; // feed into print/process
  double time_elapsed_sample_air  = NAN; // feed into print/process
  float millis_since_start        = NAN;
  float millis_since_sample_start = NAN;

  printf("a1 = %.16g\n", a1);
  printf("a2 = %.16g\n", a2);

  // start moving data to GPUs
  for (int gid = 0; gid < ngpus; gid++) {
    HostData<Real>* ghd = &(ghds[gid]);
    printf("GPU %d -- ", gid);
    printf("Nx=%ld Ns=%ld Nr=%ld Nb=%ld Nbl=%ld Nba=%ld\n", ghd->Nx, ghd->Ns, ghd->Nr, ghd->Nb, ghd->Nbl, ghd->Nba);
  }

  int64_t Ns_read  = 0;
  int64_t Nr_read  = 0;
  int64_t Nb_read  = 0;
  int64_t Nbl_read = 0;
  int64_t Nba_read = 0;
  int64_t Nx_read  = 0;
  int64_t Nx_pos   = 0;
  // uint64_t Nx_pos2=0;

  Real* u_out_buf = nullptr;
  gpuErrchk(hipHostMalloc(&u_out_buf, (size_t)(sim.Nr * sizeof(Real))));
  memset(u_out_buf, 0, (size_t)(sim.Nr * sizeof(Real))); // set floats to zero

  int64_t Nzy = (sim.Nz) * (sim.Ny); // area-slice

  // here we recalculate indices to move to devices
  for (int gid = 0; gid < ngpus; gid++) {
    gpuErrchk(hipSetDevice(gid));

    DeviceData<Real>* gd = &(gds[gid]);
    HostData<Real>* ghd  = &(ghds[gid]);
    printf("---------\n");
    printf("GPU %d\n", gid);
    printf("---------\n");

    printf("Nx to read = %ld\n", ghd->Nx);
    printf("Nb to read = %ld\n", ghd->Nb);
    printf("Nbl to read = %ld\n", ghd->Nbl);
    printf("Nba to read = %ld\n", ghd->Nba);
    printf("Ns to read = %ld\n", ghd->Ns);
    printf("Nr to read = %ld\n", ghd->Nr);

    // Nxh (effective Nx with extra halos)
    ghd->Nxh = ghd->Nx;
    if (gid > 0) {
      (ghd->Nxh)++; // add bottom halo
    }
    if (gid < ngpus - 1) {
      (ghd->Nxh)++; // add top halo
    }
    // calculate Npts for this device
    ghd->Npts = Nzy * (ghd->Nxh);
    // boundary mask
    ghd->Nbm = CU_DIV_CEIL(ghd->Npts, 8);

    printf("Nx=%ld Ns=%ld Nr=%ld Nb=%ld, Npts=%ld\n", ghd->Nx, ghd->Ns, ghd->Nr, ghd->Nb, ghd->Npts);

    // aliased pointers (to memory already allocated)
    ghd->in_sigs   = sim.in_sigs + Ns_read * sim.Nt;
    ghd->ssaf_bnl  = sim.ssaf_bnl + Nbl_read;
    ghd->adj_bn    = sim.adj_bn + Nb_read;
    ghd->mat_bnl   = sim.mat_bnl + Nbl_read;
    ghd->K_bn      = sim.K_bn + Nb_read;
    ghd->Q_bna     = sim.Q_bna + Nba_read;
    ghd->u_out     = sim.u_out + Nr_read * sim.Nt;
    ghd->u_out_buf = u_out_buf + Nr_read;

    // recalculate indices, these are associated host versions to copy over to
    // devices
    allocate_zeros((void**)&(ghd->bn_ixyz), ghd->Nb * sizeof(int64_t));
    allocate_zeros((void**)&(ghd->bnl_ixyz), ghd->Nbl * sizeof(int64_t));
    allocate_zeros((void**)&(ghd->bna_ixyz), ghd->Nba * sizeof(int64_t));
    allocate_zeros((void**)&(ghd->bn_mask), ghd->Nbm * sizeof(uint8_t));
    allocate_zeros((void**)&(ghd->in_ixyz), ghd->Ns * sizeof(int64_t));
    allocate_zeros((void**)&(ghd->out_ixyz), ghd->Nr * sizeof(int64_t));

    int64_t const offset = Nzy * Nx_pos;
    for (int64_t nb = 0; nb < (ghd->Nb); nb++) {
      int64_t const ii = sim.bn_ixyz[nb + Nb_read]; // global index
      int64_t const jj = ii - offset;               // local index
      PFFDTD_ASSERT(jj >= 0);
      PFFDTD_ASSERT(jj < ghd->Npts);
      ghd->bn_ixyz[nb] = jj;
      SET_BIT_VAL(ghd->bn_mask[jj >> 3], jj % 8, GET_BIT(sim.bn_mask[ii >> 3], ii % 8)); // set bit
    }
    for (int64_t nb = 0; nb < (ghd->Nbl); nb++) {
      int64_t const ii = sim.bnl_ixyz[nb + Nbl_read]; // global index
      int64_t const jj = ii - offset;                 // local index
      PFFDTD_ASSERT(jj >= 0);
      PFFDTD_ASSERT(jj < ghd->Npts);
      ghd->bnl_ixyz[nb] = jj;
    }

    for (int64_t nb = 0; nb < (ghd->Nba); nb++) {
      int64_t const ii = sim.bna_ixyz[nb + Nba_read]; // global index
      int64_t const jj = ii - offset;                 // local index
      PFFDTD_ASSERT(jj >= 0);
      PFFDTD_ASSERT(jj < ghd->Npts);
      ghd->bna_ixyz[nb] = jj;
    }

    for (int64_t ns = 0; ns < (ghd->Ns); ns++) {
      int64_t const ii = sim.in_ixyz[ns + Ns_read];
      int64_t const jj = ii - offset;
      PFFDTD_ASSERT(jj >= 0);
      PFFDTD_ASSERT(jj < ghd->Npts);
      ghd->in_ixyz[ns] = jj;
    }
    for (int64_t nr = 0; nr < (ghd->Nr); nr++) {
      int64_t const ii = sim.out_ixyz[nr + Nr_read];
      int64_t const jj = ii - offset;
      PFFDTD_ASSERT(jj >= 0);
      PFFDTD_ASSERT(jj < ghd->Npts);
      ghd->out_ixyz[nr] = jj;
    }

    gpuErrchk(hipMalloc(&(gd->u0), (size_t)((ghd->Npts) * sizeof(Real))));
    gpuErrchk(hipMemset(gd->u0, 0, (size_t)((ghd->Npts) * sizeof(Real))));

    gpuErrchk(hipMalloc(&(gd->u1), (size_t)((ghd->Npts) * sizeof(Real))));
    gpuErrchk(hipMemset(gd->u1, 0, (size_t)((ghd->Npts) * sizeof(Real))));

    gpuErrchk(hipMalloc(&(gd->K_bn), (size_t)(ghd->Nb * sizeof(int8_t))));
    gpuErrchk(hipMemcpy(gd->K_bn, ghd->K_bn, ghd->Nb * sizeof(int8_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(gd->ssaf_bnl), (size_t)(ghd->Nbl * sizeof(Real))));
    gpuErrchk(hipMemcpy(gd->ssaf_bnl, ghd->ssaf_bnl, ghd->Nbl * sizeof(Real), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(gd->u0b), (size_t)(ghd->Nbl * sizeof(Real))));
    gpuErrchk(hipMemset(gd->u0b, 0, (size_t)(ghd->Nbl * sizeof(Real))));

    gpuErrchk(hipMalloc(&(gd->u1b), (size_t)(ghd->Nbl * sizeof(Real))));
    gpuErrchk(hipMemset(gd->u1b, 0, (size_t)(ghd->Nbl * sizeof(Real))));

    gpuErrchk(hipMalloc(&(gd->u2b), (size_t)(ghd->Nbl * sizeof(Real))));
    gpuErrchk(hipMemset(gd->u2b, 0, (size_t)(ghd->Nbl * sizeof(Real))));

    gpuErrchk(hipMalloc(&(gd->u2ba), (size_t)(ghd->Nba * sizeof(Real))));
    gpuErrchk(hipMemset(gd->u2ba, 0, (size_t)(ghd->Nba * sizeof(Real))));

    gpuErrchk(hipMalloc(&(gd->vh1), (size_t)(ghd->Nbl * MMb * sizeof(Real))));
    gpuErrchk(hipMemset(gd->vh1, 0, (size_t)(ghd->Nbl * MMb * sizeof(Real))));

    gpuErrchk(hipMalloc(&(gd->gh1), (size_t)(ghd->Nbl * MMb * sizeof(Real))));
    gpuErrchk(hipMemset(gd->gh1, 0, (size_t)(ghd->Nbl * MMb * sizeof(Real))));

    gpuErrchk(hipMalloc(&(gd->u_out_buf), (size_t)(ghd->Nr * sizeof(Real))));
    gpuErrchk(hipMemset(gd->u_out_buf, 0, (size_t)(ghd->Nr * sizeof(Real))));

    gpuErrchk(hipMalloc(&(gd->bn_ixyz), (size_t)(ghd->Nb * sizeof(int64_t))));
    gpuErrchk(hipMemcpy(gd->bn_ixyz, ghd->bn_ixyz, (size_t)ghd->Nb * sizeof(int64_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(gd->bnl_ixyz), (size_t)(ghd->Nbl * sizeof(int64_t))));
    gpuErrchk(hipMemcpy(gd->bnl_ixyz, ghd->bnl_ixyz, (size_t)ghd->Nbl * sizeof(int64_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(gd->bna_ixyz), (size_t)(ghd->Nba * sizeof(int64_t))));
    gpuErrchk(hipMemcpy(gd->bna_ixyz, ghd->bna_ixyz, (size_t)ghd->Nba * sizeof(int64_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(gd->Q_bna), (size_t)(ghd->Nba * sizeof(int8_t))));
    gpuErrchk(hipMemcpy(gd->Q_bna, ghd->Q_bna, ghd->Nba * sizeof(int8_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(gd->out_ixyz), (size_t)(ghd->Nr * sizeof(int64_t))));
    gpuErrchk(hipMemcpy(gd->out_ixyz, ghd->out_ixyz, (size_t)ghd->Nr * sizeof(int64_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(gd->adj_bn), (size_t)(ghd->Nb * sizeof(uint16_t))));
    gpuErrchk(hipMemcpy(gd->adj_bn, ghd->adj_bn, (size_t)ghd->Nb * sizeof(uint16_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(gd->mat_bnl), (size_t)(ghd->Nbl * sizeof(int8_t))));
    gpuErrchk(hipMemcpy(gd->mat_bnl, ghd->mat_bnl, (size_t)ghd->Nbl * sizeof(int8_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(gd->mat_beta), (size_t)sim.Nm * sizeof(Real)));
    gpuErrchk(hipMemcpy(gd->mat_beta, sim.mat_beta, (size_t)sim.Nm * sizeof(Real), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(gd->mat_quads), (size_t)sim.Nm * MMb * sizeof(MatQuad<Real>)));
    gpuErrchk(
        hipMemcpy(gd->mat_quads, sim.mat_quads, (size_t)sim.Nm * MMb * sizeof(MatQuad<Real>), hipMemcpyHostToDevice)
    );

    gpuErrchk(hipMalloc(&(gd->bn_mask), (size_t)(ghd->Nbm * sizeof(uint8_t))));
    gpuErrchk(hipMemcpy(gd->bn_mask, ghd->bn_mask, (size_t)ghd->Nbm * sizeof(uint8_t), hipMemcpyHostToDevice));

    Ns_read += ghd->Ns;
    Nr_read += ghd->Nr;
    Nb_read += ghd->Nb;
    Nbl_read += ghd->Nbl;
    Nba_read += ghd->Nba;
    Nx_read += ghd->Nx;
    Nx_pos = Nx_read - 1; // back up one at subsequent stage

    printf("Nx_read = %ld\n", Nx_read);
    printf("Nb_read = %ld\n", Nb_read);
    printf("Nbl_read = %ld\n", Nbl_read);
    printf("Ns_read = %ld\n", Ns_read);
    printf("Nr_read = %ld\n", Nr_read);

    printf("Global memory allocation done\n");
    printf("\n");

    // swapping x and z here (CUDA has first dim contiguous)
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuNx), &(sim.Nz), sizeof(int64_t)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuNy), &(sim.Ny), sizeof(int64_t)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuNz), &(ghd->Nxh), sizeof(int64_t)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuNb), &(ghd->Nb), sizeof(int64_t)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuNbl), &(ghd->Nbl), sizeof(int64_t)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuNba), &(ghd->Nba), sizeof(int64_t)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuMb), sim.Mb, sim.Nm * sizeof(int8_t)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuNxNy), &Nzy,
                                 sizeof(int64_t))); // same for all devices

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(c1), &a1, sizeof(Real)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(c2), &a2, sizeof(Real)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cl), &l, sizeof(Real)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(csl2), &sl2, sizeof(Real)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(clo2), &lo2, sizeof(Real)));

    printf("Constant memory loaded\n");
    printf("\n");

    // threads grids and blocks (swap x and z)
    int64_t const cuGx  = CU_DIV_CEIL(sim.Nz - 2, cuBx);
    int64_t const cuGy  = CU_DIV_CEIL(sim.Ny - 2, cuBy);
    int64_t const cuGz  = CU_DIV_CEIL(ghd->Nxh - 2, cuBz);
    int64_t const cuGr  = CU_DIV_CEIL(ghd->Nr, cuBrw);
    int64_t const cuGb  = CU_DIV_CEIL(ghd->Nb, cuBb);
    int64_t const cuGbl = CU_DIV_CEIL(ghd->Nbl, cuBb);
    int64_t const cuGba = CU_DIV_CEIL(ghd->Nba, cuBb);

    int64_t const cuGx2 = CU_DIV_CEIL(sim.Nz, cuBx2);   // full face
    int64_t const cuGz2 = CU_DIV_CEIL(ghd->Nxh, cuBy2); // full face

    PFFDTD_ASSERT(cuGx >= 1);
    PFFDTD_ASSERT(cuGy >= 1);
    PFFDTD_ASSERT(cuGz >= 1);
    PFFDTD_ASSERT(cuGr >= 1);
    PFFDTD_ASSERT(cuGb >= 1);
    PFFDTD_ASSERT(cuGbl >= 1);
    PFFDTD_ASSERT(cuGba >= 1);

    gd->block_dim_air     = dim3(cuBx, cuBy, cuBz);
    gd->block_dim_readout = dim3(cuBrw, 1, 1);
    gd->block_dim_bn      = dim3(cuBb, 1, 1);

    gd->grid_dim_air     = dim3(cuGx, cuGy, cuGz);
    gd->grid_dim_readout = dim3(cuGr, 1, 1);
    gd->grid_dim_bn      = dim3(cuGb, 1, 1);
    gd->grid_dim_bnl     = dim3(cuGbl, 1, 1);
    gd->grid_dim_bna     = dim3(cuGba, 1, 1);

    gd->block_dim_halo_xy = dim3(cuBx2, cuBy2, 1);
    gd->block_dim_halo_yz = dim3(cuBx2, cuBy2, 1);
    gd->block_dim_halo_xz = dim3(cuBx2, cuBy2, 1);
    gd->grid_dim_halo_xy  = dim3(CU_DIV_CEIL(sim.Nz, cuBx2), CU_DIV_CEIL(sim.Ny, cuBy2), 1);
    gd->grid_dim_halo_yz  = dim3(CU_DIV_CEIL(sim.Ny, cuBx2), CU_DIV_CEIL(ghd->Nxh, cuBy2), 1);
    gd->grid_dim_halo_xz  = dim3(CU_DIV_CEIL(sim.Nz, cuBx2), CU_DIV_CEIL(ghd->Nxh, cuBy2), 1);

    gd->block_dim_fold = dim3(cuBx2, cuBy2, 1);
    gd->grid_dim_fold  = dim3(cuGx2, cuGz2, 1);

    // create streams
    gpuErrchk(hipStreamCreate(&(gd->cuStream_air)));
    gpuErrchk(hipStreamCreate(&(gd->cuStream_bn))); // no priority

    // cuda events
    gpuErrchk(hipEventCreate(&(gd->cuEv_air_start)));
    gpuErrchk(hipEventCreate(&(gd->cuEv_air_end)));
    gpuErrchk(hipEventCreate(&(gd->cuEv_bn_roundtrip_start)));
    gpuErrchk(hipEventCreate(&(gd->cuEv_bn_roundtrip_end)));
    gpuErrchk(hipEventCreate(&(gd->cuEv_readout_end)));
  }
  PFFDTD_ASSERT(Nb_read == sim.Nb);
  PFFDTD_ASSERT(Nbl_read == sim.Nbl);
  PFFDTD_ASSERT(Nba_read == sim.Nba);
  PFFDTD_ASSERT(Nr_read == sim.Nr);
  PFFDTD_ASSERT(Ns_read == sim.Ns);
  PFFDTD_ASSERT(Nx_read == sim.Nx);

  // these will be on first GPU only
  hipEvent_t cuEv_main_start        = nullptr;
  hipEvent_t cuEv_main_end          = nullptr;
  hipEvent_t cuEv_main_sample_start = nullptr;
  hipEvent_t cuEv_main_sample_end   = nullptr;
  gpuErrchk(hipSetDevice(0));
  gpuErrchk(hipEventCreate(&cuEv_main_start));
  gpuErrchk(hipEventCreate(&cuEv_main_end));
  gpuErrchk(hipEventCreate(&cuEv_main_sample_start));
  gpuErrchk(hipEventCreate(&cuEv_main_sample_end));

  for (int64_t n = 0; n < sim.Nt; n++) {    // loop over time-steps
    for (int gid = 0; gid < ngpus; gid++) { // loop over GPUs (one thread launches all kernels)
      gpuErrchk(hipSetDevice(gid));
      DeviceData<Real>* gd = &(gds[gid]);  // get struct of device pointers
      HostData<Real>* ghd  = &(ghds[gid]); // get struct of host points (corresponding to device)

      // start first timer
      if (gid == 0) {
        if (n == 0) {
          // not sure if to put on stream, check slides again
          gpuErrchk(hipEventRecord(cuEv_main_start, nullptr));
        }
        gpuErrchk(hipEventRecord(cuEv_main_sample_start, nullptr));
      }
      // boundary updates (using intermediate buffer)
      gpuErrchk(hipEventRecord(gd->cuEv_bn_roundtrip_start, gd->cuStream_bn));

      // boundary updates
      if (sim.fcc_flag == 0) {
        KernelBoundaryRigidCart<<<gd->grid_dim_bn, gd->block_dim_bn, 0, gd->cuStream_bn>>>(
            gd->u0,
            gd->u1,
            gd->adj_bn,
            gd->bn_ixyz,
            gd->K_bn
        );
      } else {
        KernelFoldFCC<<<gd->grid_dim_fold, gd->block_dim_fold, 0, gd->cuStream_bn>>>(gd->u1);
        KernelBoundaryRigidFCC<<<gd->grid_dim_bn, gd->block_dim_bn, 0, gd->cuStream_bn>>>(
            gd->u0,
            gd->u1,
            gd->adj_bn,
            gd->bn_ixyz,
            gd->K_bn
        );
      }
      // using buffer to then update FD boundaries
      CopyFromGridKernel<<<gd->grid_dim_bnl, gd->block_dim_bn, 0, gd->cuStream_bn>>>(
          gd->u0b,
          gd->u0,
          gd->bnl_ixyz,
          ghd->Nbl
      );
      // possible this could be moved to host
      KernelBoundaryFD<<<gd->grid_dim_bnl, gd->block_dim_bn, 0, gd->cuStream_bn>>>(
          gd->u0b,
          gd->u2b,
          gd->vh1,
          gd->gh1,
          gd->ssaf_bnl,
          gd->mat_bnl,
          gd->mat_beta,
          gd->mat_quads
      );
      // copy to back to grid
      CopyToGridKernel<<<gd->grid_dim_bnl, gd->block_dim_bn, 0, gd->cuStream_bn>>>(
          gd->u0,
          gd->u0b,
          gd->bnl_ixyz,
          ghd->Nbl
      );
      gpuErrchk(hipEventRecord(gd->cuEv_bn_roundtrip_end, gd->cuStream_bn));

      // air updates (including source
      gpuErrchk(hipStreamWaitEvent(gd->cuStream_air, gd->cuEv_bn_roundtrip_end,
                                    0)); // might as well wait
      // run air kernel (with mask)
      gpuErrchk(hipEventRecord(gd->cuEv_air_start, gd->cuStream_air));

      // for absorbing boundaries at boundaries of grid
      CopyFromGridKernel<<<gd->grid_dim_bna, gd->block_dim_bn, 0, gd->cuStream_air>>>(
          gd->u2ba,
          gd->u0,
          gd->bna_ixyz,
          ghd->Nba
      );
      if (gid == 0) {
        FlipHaloXY_Zbeg<<<gd->grid_dim_halo_xy, gd->block_dim_halo_xy, 0, gd->cuStream_air>>>(gd->u1);
      }
      if (gid == ngpus - 1) {
        FlipHaloXY_Zend<<<gd->grid_dim_halo_xy, gd->block_dim_halo_xy, 0, gd->cuStream_air>>>(gd->u1);
      }
      FlipHaloXZ_Ybeg<<<gd->grid_dim_halo_xz, gd->block_dim_halo_xz, 0, gd->cuStream_air>>>(gd->u1);
      if (sim.fcc_flag == 0) {
        FlipHaloXZ_Yend<<<gd->grid_dim_halo_xz, gd->block_dim_halo_xz, 0, gd->cuStream_air>>>(gd->u1);
      }
      FlipHaloYZ_Xbeg<<<gd->grid_dim_halo_yz, gd->block_dim_halo_yz, 0, gd->cuStream_air>>>(gd->u1);
      FlipHaloYZ_Xend<<<gd->grid_dim_halo_yz, gd->block_dim_halo_yz, 0, gd->cuStream_air>>>(gd->u1);

      // injecting source first, negating sample to add it in first (NB source
      // on different stream than bn)
      for (int64_t ns = 0; ns < ghd->Ns; ns++) {
        AddIn<<<1, 1, 0, gd->cuStream_air>>>(gd->u0 + ghd->in_ixyz[ns], (Real)(-(ghd->in_sigs[ns * sim.Nt + n])));
      }
      // now air updates (not conflicting with bn updates because of bn_mask)
      if (sim.fcc_flag == 0) {
        KernelAirCart<<<gd->grid_dim_air, gd->block_dim_air, 0, gd->cuStream_air>>>(gd->u0, gd->u1, gd->bn_mask);
      } else {
        KernelAirFCC<<<gd->grid_dim_air, gd->block_dim_air, 0, gd->cuStream_air>>>(gd->u0, gd->u1, gd->bn_mask);
      }
      // boundary ABC loss
      KernelBoundaryABC<<<gd->grid_dim_bna, gd->block_dim_bn, 0, gd->cuStream_air>>>(
          gd->u0,
          gd->u2ba,
          gd->Q_bna,
          gd->bna_ixyz
      );
      gpuErrchk(hipEventRecord(gd->cuEv_air_end, gd->cuStream_air)); // for timing

      // readouts
      CopyFromGridKernel<<<gd->grid_dim_readout, gd->block_dim_readout, 0, gd->cuStream_bn>>>(
          gd->u_out_buf,
          gd->u1,
          gd->out_ixyz,
          ghd->Nr
      );
      // then async memory copy of outputs (not really async because on same
      // stream as CopyFromGridKernel)
      gpuErrchk(hipMemcpyAsync(
          ghd->u_out_buf,
          gd->u_out_buf,
          ghd->Nr * sizeof(Real),
          hipMemcpyDeviceToHost,
          gd->cuStream_bn
      ));
      gpuErrchk(hipEventRecord(gd->cuEv_readout_end, gd->cuStream_bn));
    }

    // readouts
    for (int gid = 0; gid < ngpus; gid++) {
      gpuErrchk(hipSetDevice(gid));
      DeviceData<Real>* gd = &(gds[gid]);
      HostData<Real>* ghd  = &(ghds[gid]);
      gpuErrchk(hipEventSynchronize(gd->cuEv_readout_end));
      // copy grid points off output buffer
      for (int64_t nr = 0; nr < ghd->Nr; nr++) {
        ghd->u_out[nr * sim.Nt + n] = (double)(ghd->u_out_buf[nr]);
      }
    }
    // synchronise streams
    for (int gid = 0; gid < ngpus; gid++) {
      gpuErrchk(hipSetDevice(gid));
      DeviceData<Real>* gd = &(gds[gid]);                 // don't really need to set gpu device to sync
      gpuErrchk(hipStreamSynchronize(gd->cuStream_air)); // interior complete
      gpuErrchk(hipStreamSynchronize(gd->cuStream_bn));  // transfer complete
    }
    // dst then src, stream with src gives best performance (CUDA thing)

    // now asynchronous halo swaps, even/odd pairs concurrent
    // these are not async to rest of scheme, just async to other swaps

    // copy forward (even)
    for (int gid = 0; gid < ngpus - 1; gid += 2) {
      gpuErrchk(hipSetDevice(gid));
      gpuErrchk(hipMemcpyPeerAsync(
          gds[gid + 1].u0,
          gid + 1,
          gds[gid].u0 + Nzy * (ghds[gid].Nxh - 2),
          gid,
          (size_t)(Nzy * sizeof(Real)),
          gds[gid].cuStream_bn
      ));
    }
    // copy back (odd)
    for (int gid = 1; gid < ngpus; gid += 2) {
      gpuErrchk(hipSetDevice(gid));
      gpuErrchk(hipMemcpyPeerAsync(
          gds[gid - 1].u0 + Nzy * (ghds[gid - 1].Nxh - 1),
          gid - 1,
          gds[gid].u0 + Nzy,
          gid,
          (size_t)(Nzy * sizeof(Real)),
          gds[gid].cuStream_bn
      ));
    }
    // copy forward (odd)
    for (int gid = 1; gid < ngpus - 1; gid += 2) {
      gpuErrchk(hipSetDevice(gid));
      gpuErrchk(hipMemcpyPeerAsync(
          gds[gid + 1].u0,
          gid + 1,
          gds[gid].u0 + Nzy * (ghds[gid].Nxh - 2),
          gid,
          (size_t)(Nzy * sizeof(Real)),
          gds[gid].cuStream_bn
      ));
    }
    // copy back (even) -- skip zero
    for (int gid = 2; gid < ngpus; gid += 2) {
      gpuErrchk(hipSetDevice(gid));
      gpuErrchk(hipMemcpyPeerAsync(
          gds[gid - 1].u0 + Nzy * (ghds[gid - 1].Nxh - 1),
          gid - 1,
          gds[gid].u0 + Nzy,
          gid,
          (size_t)(Nzy * sizeof(Real)),
          gds[gid].cuStream_bn
      ));
    }

    for (int gid = 0; gid < ngpus; gid++) {
      gpuErrchk(hipSetDevice(gid));
      DeviceData<Real>* gd = &(gds[gid]);
      gpuErrchk(hipStreamSynchronize(gd->cuStream_bn)); // transfer complete
    }
    for (int gid = 0; gid < ngpus; gid++) {
      DeviceData<Real>* gd = &(gds[gid]);
      // update pointers
      Real* tmp_ptr = nullptr;
      tmp_ptr       = gd->u1;
      gd->u1        = gd->u0;
      gd->u0        = tmp_ptr;

      // will use extra vector for this (simpler than extra copy kernel)
      tmp_ptr = gd->u2b;
      gd->u2b = gd->u1b;
      gd->u1b = gd->u0b;
      gd->u0b = tmp_ptr;

      if (gid == 0) {
        gpuErrchk(hipSetDevice(gid));
        gpuErrchk(hipEventRecord(cuEv_main_sample_end, nullptr));
      }
    }

    {
      // timing only on gpu0
      gpuErrchk(hipSetDevice(0));
      DeviceData<Real>* gd = &(gds[0]);
      gpuErrchk(hipEventSynchronize(cuEv_main_sample_end)); // not sure this is correct
      gpuErrchk(hipEventElapsedTime(&millis_since_start, cuEv_main_start, cuEv_main_sample_end));
      gpuErrchk(hipEventElapsedTime(&millis_since_sample_start, cuEv_main_sample_start, cuEv_main_sample_end));

      time_elapsed        = millis_since_start / 1000;
      time_elapsed_sample = millis_since_sample_start / 1000;

      float millis_air = NAN;
      float millis_bn  = NAN;
      gpuErrchk(hipEventElapsedTime(&millis_air, gd->cuEv_air_start, gd->cuEv_air_end));
      time_elapsed_sample_air = 0.001 * millis_air;
      time_elapsed_air += time_elapsed_sample_air;

      // not full picutre, only first gpu
      gpuErrchk(hipEventElapsedTime(&millis_bn, gd->cuEv_bn_roundtrip_start, gd->cuEv_bn_roundtrip_end));

      time_elapsed_sample_bn = millis_bn / 1000.0;
      time_elapsed_bn += time_elapsed_sample_bn;

      print_progress(
          n,
          sim.Nt,
          sim.Npts,
          sim.Nb,
          time_elapsed,
          time_elapsed_sample,
          time_elapsed_air,
          time_elapsed_sample_air,
          time_elapsed_bn,
          time_elapsed_sample_bn,
          ngpus
      );
    }
  }
  printf("\n");

  for (int gid = 0; gid < ngpus; gid++) {
    gpuErrchk(hipSetDevice(gid));
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
  }
  {
    // timing (on device 0)
    gpuErrchk(hipSetDevice(0));
    gpuErrchk(hipEventRecord(cuEv_main_end));
    gpuErrchk(hipEventSynchronize(cuEv_main_end));

    gpuErrchk(hipEventElapsedTime(&millis_since_start, cuEv_main_start, cuEv_main_end));
    time_elapsed = millis_since_start / 1000;
  }

  /*------------------------
   * FREE WILLY
  ------------------------*/
  gpuErrchk(hipSetDevice(0));
  gpuErrchk(hipEventDestroy(cuEv_main_start));
  gpuErrchk(hipEventDestroy(cuEv_main_end));
  gpuErrchk(hipEventDestroy(cuEv_main_sample_start));
  gpuErrchk(hipEventDestroy(cuEv_main_sample_end));
  for (int gid = 0; gid < ngpus; gid++) {
    gpuErrchk(hipSetDevice(gid));
    DeviceData<Real>* gd = &(gds[gid]);
    HostData<Real>* ghd  = &(ghds[gid]);
    // cleanup streams
    gpuErrchk(hipStreamDestroy(gd->cuStream_air));
    gpuErrchk(hipStreamDestroy(gd->cuStream_bn));

    // cleanup events
    gpuErrchk(hipEventDestroy(gd->cuEv_air_start));
    gpuErrchk(hipEventDestroy(gd->cuEv_air_end));
    gpuErrchk(hipEventDestroy(gd->cuEv_bn_roundtrip_start));
    gpuErrchk(hipEventDestroy(gd->cuEv_bn_roundtrip_end));
    gpuErrchk(hipEventDestroy(gd->cuEv_readout_end));

    // free memory
    gpuErrchk(hipFree(gd->u0));
    gpuErrchk(hipFree(gd->u1));
    gpuErrchk(hipFree(gd->out_ixyz));
    gpuErrchk(hipFree(gd->bn_ixyz));
    gpuErrchk(hipFree(gd->bnl_ixyz));
    gpuErrchk(hipFree(gd->bna_ixyz));
    gpuErrchk(hipFree(gd->Q_bna));
    gpuErrchk(hipFree(gd->adj_bn));
    gpuErrchk(hipFree(gd->mat_bnl));
    gpuErrchk(hipFree(gd->K_bn));
    gpuErrchk(hipFree(gd->ssaf_bnl));
    gpuErrchk(hipFree(gd->mat_beta));
    gpuErrchk(hipFree(gd->mat_quads));
    gpuErrchk(hipFree(gd->bn_mask));
    gpuErrchk(hipFree(gd->u0b));
    gpuErrchk(hipFree(gd->u1b));
    gpuErrchk(hipFree(gd->u2b));
    gpuErrchk(hipFree(gd->u2ba));
    gpuErrchk(hipFree(gd->vh1));
    gpuErrchk(hipFree(gd->gh1));
    gpuErrchk(hipFree(gd->u_out_buf));
    free(ghd->bn_mask);
    free(ghd->bn_ixyz);
    free(ghd->bnl_ixyz);
    free(ghd->bna_ixyz);
    free(ghd->in_ixyz);
    free(ghd->out_ixyz);
  }
  gpuErrchk(hipHostFree(u_out_buf));
  free(gds);
  free(ghds);

  // reset after frees (for some reason it conflicts with hipHostFree)
  for (int gid = 0; gid < ngpus; gid++) {
    gpuErrchk(hipSetDevice(gid));
    gpuErrchk(hipDeviceReset());
  }

  printf("Boundary loop: %.6fs, %.2f Mvox/s\n", time_elapsed_bn, sim.Nb * sim.Nt / 1e6 / time_elapsed_bn);
  printf("Air update: %.6fs, %.2f Mvox/s\n", time_elapsed_air, sim.Npts * sim.Nt / 1e6 / time_elapsed_air);
  printf("Combined (total): %.6fs, %.2f Mvox/s\n", time_elapsed, sim.Npts * sim.Nt / 1e6 / time_elapsed);
  return time_elapsed;
}

} // namespace pffdtd
